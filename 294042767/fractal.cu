#include "hip/hip_runtime.h"
/*
Computing a movie of zooming into a fractal using Cuda

Author: Jonathan Ma
*/

#include <stdlib.h>
#include <stdio.h>
#include "timer.h"
#include "fractal.h"


__constant__ double Delta;
__constant__ double xMid;
__constant__ double yMid;
__constant__ int height;
__constant__ int width;
__constant__ double aspect_ratio;


__global__ 
void cuda_hello(){
        printf("Hello World from GPU!\n");
}

__global__ 
void compute_fractal(unsigned char *pic, int num_frames) {
  extern __shared__ unsigned char shared_frame[];
  int frame = blockIdx.z;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row >= height || col >= width || frame >= num_frames) return;
  const double delta = Delta*pow(0.98, frame);
  const double x0 = xMid - delta * aspect_ratio;
  const double y0 = yMid - delta;
  const double dx = 2.0 * delta * aspect_ratio / width;
  const double dy = 2.0 * delta / height;

  const double cy = y0 + row * dy;
  const double cx = x0 + col * dx;

  double x = cx;
  double y = cy;
  int depth = 256;

  double x2, y2;
  do {
      x2 = x * x;
      y2 = y * y;
      y = 2 * x * y + cy;
      x = x2 - y2 + cx;
      depth--;
  } while ((depth > 0) && ((x2 + y2) < 5.0));
  int local_index = threadIdx.y * blockDim.x + threadIdx.x;
  shared_frame[local_index] = (unsigned char)depth;
  __syncthreads();
  if (row < height && col < width) {
      pic[frame * height * width + row * width + col] = shared_frame[local_index];
  }
}

int main(int argc, char *argv[]) {
  double start, end;

  printf("Fractal v1.0 [cuda]\n");

  /* read command line arguments */
  if (argc != 4) {fprintf(stderr, "usage: %s height width num_frames\n", argv[0]); exit(-1);}
  int hst_width = atoi(argv[1]);
  if (hst_width < 10) {fprintf(stderr, "error: width must be at least 10\n"); exit(-1);}
  int hst_height = atoi(argv[2]);
  if (hst_height < 10) {fprintf(stderr, "error: height must be at least 10\n"); exit(-1);}
  int num_frames = atoi(argv[3]);
  if (num_frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
  printf("Computing %d frames of %d by %d fractal\n", num_frames, hst_width, hst_height);

  /* allocate image array */
  int sz = num_frames * hst_height * hst_width * sizeof(unsigned char);

  unsigned char *pic = (unsigned char*)malloc(sz);
  unsigned char *dev_pic;

  int malloc_status = hipMalloc((void**) &dev_pic, sz);
  if(malloc_status == 0){
      printf("Successfully allocated device memory.\n");
  }
  if(malloc_status == 100){
      printf("No Cuda capable devices found.\n");
  }

  /* start time */
  GET_TIME(start);

  /* compute frames */
  const double hst_aspect_ratio = (double)hst_width/hst_height;

  hipMemcpy(dev_pic, pic, sz, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(aspect_ratio), &hst_aspect_ratio, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(height), &hst_height, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(width), &hst_width, sizeof(int));

  static const double hst_Delta = 0.001;
  static const double hst_xMid =  0.23701;
  static const double hst_yMid =  0.521;

  hipMemcpyToSymbol(HIP_SYMBOL(Delta), &hst_Delta, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(xMid), &hst_xMid, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(yMid), &hst_yMid, sizeof(double));


  dim3 blockSize(32,32);
  dim3 gridSize((hst_width + blockSize.x - 1) / blockSize.x,
          (hst_height + blockSize.y - 1) / blockSize.y, 
          num_frames);
  int sharedMemSize = blockSize.x * blockSize.y * sizeof(unsigned char);
  compute_fractal<<<gridSize, blockSize, sharedMemSize>>>(dev_pic, num_frames);
  hipDeviceSynchronize();
  hipMemcpy(pic, dev_pic, sz, hipMemcpyDeviceToHost);

  /* end time */
  GET_TIME(end);
  double elapsed = end - start;
  printf("Cuda compute time: %.4f s\n", elapsed);
  /* write frames to BMP files */
  if ((hst_width <= 320) && (num_frames <= 10)) { /* do not write if images large or many */
    for (int frame = 0; frame < num_frames; frame++) {
      char name[32];
      sprintf(name, "fractal%d.bmp", frame + 1000);
      writeBMP(hst_width, hst_height, &pic[frame * hst_height * hst_width], name);
    }
  }

  hipFree(dev_pic);
  free(pic);
  return 0;
} /* main */
