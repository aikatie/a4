#include "hip/hip_runtime.h"
/*
Computing a movie of zooming into a fractal

Original C++ code by Martin Burtscher, Texas State University

Reference: E. Ayguade et al., 
           "Peachy Parallel Assignments (EduHPC 2018)".
           2018 IEEE/ACM Workshop on Education for High-Performance Computing (EduHPC), pp. 78-85,
           doi: 10.1109/EduHPC.2018.00012

Copyright (c) 2018, Texas State University. All rights reserved.

Redistribution and usage in source and binary form, with or without
modification, is only permitted for educational use.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <stdlib.h>
#include <stdio.h>
#include "timer.h"
#include "fractal.h"

static const float Delta = 0.001;
static const float xMid =  0.23701;
static const float yMid =  0.521;

__global__ void cuda_internal(int height, int width, float aspect_ratio, float x0, float y0, float dx, float dy, float delta, unsigned char* pic, int frame)
{
	for (int row = 0; row < height; row++) 
	{
		const float cy = y0 + row * dy;
		for (int col = 0; col < width; col++) 
		{
			const float cx = x0 + col * dx;
			
			float x = cx;
			float y = cy;
			int depth = 256;
			float x2 = 0.0;
			float y2 = 0.0;
			while ((depth > 0) && ((x2 + y2) < 5.0))
			{
				x2 = x * x;
				y2 = y * y;
				y = 2 * x * y + cy;
				x = x2 - y2 + cx;
				depth--;
			}
			pic[frame * height * width + row * width + col] = (unsigned char)depth;
		}
	}
	
}


int main(int argc, char *argv[]) 
{
	float start, end;

	printf("Fractal v1.6 [parallel]\n");

	/* read command line arguments */
	if (argc != 4) 
	{
		fprintf(stderr, "usage: %s height width num_frames\n", argv[0]); exit(-1);
	}
	int width = atoi(argv[1]);
	if (width < 10) 
	{
		fprintf(stderr, "error: width must be at least 10\n"); exit(-1);
	}
	int height = atoi(argv[2]);
	if (height < 10) 
	{
		fprintf(stderr, "error: height must be at least 10\n"); exit(-1);
	}
	int num_frames = atoi(argv[3]);
	if (num_frames < 1) 
	{
		fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);
	}
	printf("Computing %d frames of %d by %d fractal\n", num_frames, width, height);

	/* allocate image array */
	unsigned char *pic = (unsigned char*)malloc(num_frames * height * width * sizeof(unsigned char));

	/* start time */
	GET_TIME(start);

	/* compute frames */
	const float aspect_ratio = (float)width/height;
	float delta = Delta;
	for (int frame = 0; frame < num_frames; frame++) 
	{

		const float x0 = xMid - delta * aspect_ratio;
		const float y0 = yMid - delta;
		const float dx = 2.0 * delta * aspect_ratio / width;
		const float dy = 2.0 * delta / height;
		
		cuda_internal<<<height, width>>>(height, width, aspect_ratio, x0, y0, dx, dy, delta, pic, frame);
		
		/*
		for (int row = 0; row < height; row++) 
		{

			const float cy = y0 + row * dy;
			for (int col = 0; col < width; col++) 
			{
				const float cx = x0 + col * dx;
				float x = cx;
				float y = cy;
				
				int depth = 256;
				float x2 = 0.0;
				float y2 = 0.0;
				while (depth > 0) && ((x2 + y2) < 5.0)
				{
					x2 = x * x;
					y2 = y * y;
					y = 2 * x * y + cy;
					x = x2 - y2 + cx;
					depth--;
				}
				pic[frame * height * width + row * width + col] = (unsigned char)depth;
			}
		}
		*/
		delta *= 0.98;
	}

	/* end time */
	GET_TIME(end);
	float elapsed = end - start;
	printf("Parallel compute time: %.4f s\n", elapsed);

	/* write frames to BMP files */
	if ((width <= 320) && (num_frames <= 100)) 
	{ 
		/* do not write if images large or many */
		for (int frame = 0; frame < num_frames; frame++) 
		{
			char name[32];
			sprintf(name, "fractal%d.bmp", frame + 1000);
			writeBMP(width, height, &pic[frame * height * width], name);
		}
	}

	free(pic);

	return 0;
} /* main */

