#include "hip/hip_runtime.h"
//To calculate speedup, uncomment the serial implementation and print statements for printing the serial time and speedup

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "timer.h"
#include "fractal.h"

static const double Delta = 0.001;
static const double xMid = 0.23701;
static const double yMid = 0.521;

__global__ void fractalKernel(unsigned char *pic, int width, int height, int num_frames, double delta, double aspect_ratio) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int frame = blockIdx.z; 

    __shared__ unsigned char sharedDepth[128][128];

     if (row < height && col < width) {
        double current_delta = delta * pow(0.98, frame);
        double x0 = xMid - current_delta * aspect_ratio;
        double y0 = yMid - current_delta;
        double dx = 2.0 * current_delta * aspect_ratio / width;
        double dy = 2.0 * current_delta / height;

        double cx = x0 + col * dx;
        double cy = y0 + row * dy;
        double x = cx;
        double y = cy;
        int depth = 256;

        double x2, y2;
        do {
            x2 = x * x;
            y2 = y * y;
            y = 2 * x * y + cy;
            x = x2 - y2 + cx;
            depth--;
        } while ((depth > 0) && ((x2 + y2) < 5.0));
        sharedDepth[threadIdx.y][threadIdx.x] = (unsigned char)depth;
        __syncthreads();

        int idx = frame * height * width + row * width + col;
        pic[idx] = sharedDepth[threadIdx.y][threadIdx.x];
    }
}

int main(int argc, char *argv[]) {
    if (argc != 4) {
        fprintf(stderr, "Usage: %s height width num_frames\n", argv[0]);
        exit(-1);
    }

    if (argc != 4) {fprintf(stderr, "usage: %s height width num_frames\n", argv[0]); exit(-1);}
    int width = atoi(argv[1]);
    if (width < 10) {fprintf(stderr, "error: width must be at least 10\n"); exit(-1);}
    int height = atoi(argv[2]);
    if (height < 10) {fprintf(stderr, "error: height must be at least 10\n"); exit(-1);}
    int num_frames = atoi(argv[3]);
    if (num_frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
    printf("Computing %d frames of %d by %d fractal\n", num_frames, width, height);

    unsigned char *pic_serial = (unsigned char *)malloc(num_frames * height * width * sizeof(unsigned char));
    unsigned char *pic_cuda = (unsigned char *)malloc(num_frames * height * width * sizeof(unsigned char));

    // serial imp
    double start_serial, end_serial;
    GET_TIME(start_serial);

    double delta = Delta;
    const double aspect_ratio = (double)width / height;

 /*   for (int frame = 0; frame < num_frames; frame++) {
        const double x0 = xMid - delta * aspect_ratio;
        const double y0 = yMid - delta;
        const double dx = 2.0 * delta * aspect_ratio / width;
        const double dy = 2.0 * delta / height;

        for (int row = 0; row < height; row++) {
            const double cy = y0 + row * dy;
            for (int col = 0; col < width; col++) {
                const double cx = x0 + col * dx;
                double x = cx;
                double y = cy;
                int depth = 256;
                double x2, y2;
                do {
                    x2 = x * x;
                    y2 = y * y;
                    y = 2 * x * y + cy;
                    x = x2 - y2 + cx;
                    depth--;
                } while ((depth > 0) && ((x2 + y2) < 5.0));

                pic_serial[frame * height * width + row * width + col] = (unsigned char)depth;
            }
        }
        delta *= 0.98;
    }
*/
    GET_TIME(end_serial);
    double elapsed_serial = end_serial - start_serial;
//    printf("Serial compute time: %.4f s\n", elapsed_serial);

    // cuda imp
    double start_cuda, end_cuda;
    unsigned char *d_pic;
    hipMalloc(&d_pic, num_frames * height * width * sizeof(unsigned char));
    hipError_t err = hipMalloc(&d_pic, num_frames * height * width * sizeof(unsigned char));
    if (err != hipSuccess) {
        fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(err));
        return -1;
    }
    GET_TIME(start_cuda);
    dim3 threadsPerBlock(128, 128);
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   num_frames);
    fractalKernel<<<numBlocks, threadsPerBlock>>>(d_pic, width, height, num_frames, Delta, aspect_ratio);
    hipDeviceSynchronize();
    hipMemcpy(pic_cuda, d_pic, num_frames * height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);

    GET_TIME(end_cuda);
    double elapsed_cuda = end_cuda - start_cuda;
    printf("Parallel compute time: %.6f s\n", elapsed_cuda);

    if ((width <= 10000) && (num_frames <= 100)) {
        for (int frame = 0; frame < num_frames; frame++) {
            char name[32];
            sprintf(name, "fractal%d.bmp", frame + 1000);
            writeBMP(width, height, &pic_cuda[frame * height * width], name);
        }
    }
//    printf("Speedup: %.4f\n",elapsed_serial/elapsed_cuda);
    free(pic_serial);
    free(pic_cuda);
    hipFree(d_pic);

    return 0;
}
