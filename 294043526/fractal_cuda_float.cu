#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "timer.h"
#include "fractal.h"

static const float Delta = 0.001f;
static const float xMid =  0.23701f;
static const float yMid =  0.521f;

__global__ void computeFractal(int width, int height, float delta, float aspect_ratio, unsigned char *pic, int frame) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < height && col < width) {
        const float x0 = xMid - delta * aspect_ratio;
        const float y0 = yMid - delta;
        const float dx = 2.0f * delta * aspect_ratio / width;
        const float dy = 2.0f * delta / height;
        
        const float cy = y0 + row * dy;
        const float cx = x0 + col * dx;
        
        float x = cx;
        float y = cy;
        int depth = 256;
        
        float x2, y2;
        do {
            x2 = x * x;
            y2 = y * y;
            y = 2 * x * y + cy;
            x = x2 - y2 + cx;
            depth--;
        } while ((depth > 0) && ((x2 + y2) < 5.0f));
        
        pic[frame * height * width + row * width + col] = (unsigned char)depth;
    }
}

int main(int argc, char *argv[]) {
    double start, end;
    
    printf("Fractal v1.6 [CUDA] - Using floats for computation and doubles for time tracking\n");

    /* read command line arguments */
    if (argc != 4) {
        fprintf(stderr, "usage: %s height width num_frames\n", argv[0]);
        exit(-1);
    }
    
    int width = atoi(argv[1]);
    if (width < 10) {
        fprintf(stderr, "error: width must be at least 10\n");
        exit(-1);
    }
    
    int height = atoi(argv[2]);
    if (height < 10) {
        fprintf(stderr, "error: height must be at least 10\n");
        exit(-1);
    }
    
    int num_frames = atoi(argv[3]);
    if (num_frames < 1) {
        fprintf(stderr, "error: num_frames must be at least 1\n");
        exit(-1);
    }
    
    printf("Computing %d frames of %d by %d fractal\n", num_frames, width, height);
    unsigned char *pic;
    hipMallocManaged(&pic, num_frames * height * width * sizeof(unsigned char));

    /* start time */
    GET_TIME(start);

    /* compute frames */
    const float aspect_ratio = (float)width / height;
    float delta = Delta;
        dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + 15) / 16, (height + 15) / 16);
    
    for (int frame = 0; frame < num_frames; frame++) {
        computeFractal<<<numBlocks, threadsPerBlock>>>(width, height, delta, aspect_ratio, pic, frame);
        
        hipDeviceSynchronize();
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(error));
            return -1;
        }

        delta *= 0.98f;
    }

    /* end time */
    GET_TIME(end);
    double elapsed = end - start;
    printf("CUDA compute time: %.6f s\n", elapsed);  // Using double precision for time

    /* write frames to BMP files */
    if ((width <= 320) && (num_frames <= 100)) {
        for (int frame = 0; frame < num_frames; frame++) {
            char name[32];
            sprintf(name, "fractal%d.bmp", frame + 1000);
            writeBMP(width, height, &pic[frame * height * width], name);
        }
    }

    hipFree(pic);
    
    return 0;
} /* main */
