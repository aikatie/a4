#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "timer.h"
#include "fractal.h"
#include <hip/hip_runtime.h>

static const float Delta = 0.001;
static const float xMid = 0.23701;
static const float yMid = 0.521;

// CUDA kernel for computing fractal
__global__ void computeFractal(unsigned char* pic, int width, int height, int num_frames,
                              float x0_first, float y0_first, float aspect_ratio) {
    
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int frame = blockIdx.z;


    if (col < width && row < height && frame < num_frames) {
        float delta = Delta * pow(0.98, frame);
        
        float x0 = xMid - delta * aspect_ratio;
        float y0 = yMid - delta;
        float dx = 2.0 * delta * aspect_ratio / width;
        float dy = 2.0 * delta / height;

        float cx = x0 + col * dx;
        float cy = y0 + row * dy;
        int depth = 256;
        float x2, y2;

        float x = cx;
        float y = cy; 
        
        
        do {
            x2 = x * x;
            y2 = y * y;
            y = 2 * x * y + cy;
            x = x2 - y2 + cx;
            depth--;
        } while ((depth > 0) && ((x2 + y2) < 5.0));
    
    // Store result
        pic[frame * height * width + row * width + col] = (unsigned char)depth;
    }
}

int main(int argc, char *argv[]) {
    double start, end;
    
    printf("Fractal v1.6 [CUDA]\n");
    
    // Check command line arguments
    if (argc != 4) {
        fprintf(stderr, "usage: %s height width num_frames\n", argv[0]);
        exit(-1);
    }
    int width = atoi(argv[1]);
    if (width < 10) {
        fprintf(stderr, "error: width must be at least 10\n");
        exit(-1);
    }
    int height = atoi(argv[2]);
    if (height < 10) {
        fprintf(stderr, "error: height must be at least 10\n");
        exit(-1);
    }
    int num_frames = atoi(argv[3]);
    if (num_frames < 1) {
        fprintf(stderr, "error: num_frames must be at least 1\n");
        exit(-1);
    }
    printf("Computing %d frames of %d by %d fractal\n", num_frames, width, height);
    
    // Allocate host and device memory
    unsigned char *h_pic = (unsigned char*)malloc(num_frames * height * width * sizeof(unsigned char));
    unsigned char *d_pic;
    hipMalloc(&d_pic, num_frames * height * width * sizeof(unsigned char));
    
    // Calculate initial values
    const float aspect_ratio = (float)width / height;
    const float x0_first = xMid - Delta * aspect_ratio;
    const float y0_first = yMid - Delta;
    
    dim3 threadsPerBlock(16, 16);  // 256 threads per block
    dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (height + threadsPerBlock.y - 1) / threadsPerBlock.y,
                   num_frames
    );
    
    GET_TIME(start);
    
    // Launch kernel
    computeFractal<<<numBlocks, threadsPerBlock>>>(d_pic, width, height, num_frames, x0_first, y0_first, aspect_ratio);
    
    
    GET_TIME(end);

    hipMemcpy(h_pic, d_pic, num_frames * height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
    double elapsed = end - start;
    printf("CUDA compute time: %.6f s\n", elapsed);
    
    // Write frames to BMP files
    if ((width <= 1024) && (num_frames <= 100)) {
        for (int frame = 0; frame < num_frames; frame++) {
            char name[32];
            sprintf(name, "./fractal%d.bmp", frame + 1000);
            writeBMP(width, height, &h_pic[frame * height * width], name);
        }
    }
    
    // Free memory
    free(h_pic);
    hipFree(d_pic);
    
    return 0;
}