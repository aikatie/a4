#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "timer.h"
#include "fractal.h"

static const double Delta = 0.001;
static const double xMid =  0.23701;
static const double yMid =  0.521;

__global__ void fractal_kernel(unsigned char *pic, int width, int height, int num_frames, const double aspect_ratio, double delta){
        //something like this
        //frame represents frame of the num_frames for the thread
        //row represents row of the height for the thread
        //col represents col of the width for the thread
        int frame = blockIdx.x * blockDim.x + threadIdx.x;
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.z * blockDim.z + threadIdx.z;

        //variables from the first and second loops
        const double x0 = xMid - delta * aspect_ratio;
        const double y0 = yMid - delta;
        const double dx = 2.0 * delta * aspect_ratio / width;
        const double dy = 2.0 * delta / height;
        const double cy = y0 + row * dy;

        //body of the third nested loop
        const double cx = x0 + col * dx;
        double x = cx;
        double y = cy;
        int depth = 256;
        double x2, y2;
        do {
                x2 = x * x;
                y2 = y * y;
                y = 2 * x * y + cy;
                x = x2 - y2 + cx;
                depth--;
        } while ((depth > 0) && ((x2 + y2) < 5.0));
        pic[frame * height * width + row * width + col] = (unsigned char)depth;

}

int main(int argc, char *argv[]) {
        double start, end;

        printf("Fractal v1.6 [serial]\n");

        /* read command line arguments */
        if (argc != 4) {fprintf(stderr, "usage: %s height width num_frames\n", argv[0]); exit(-1);}
        int width = atoi(argv[1]);
        if (width < 10) {fprintf(stderr, "error: width must be at least 10\n"); exit(-1);}
        int height = atoi(argv[2]);
        if (height < 10) {fprintf(stderr, "error: height must be at least 10\n"); exit(-1);}
        int num_frames = atoi(argv[3]);
        if (num_frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
        printf("Computing %d frames of %d by %d fractal\n", num_frames, width, height);

        /* allocate image array */
        unsigned char *pic;
        hipMalloc(&pic, num_frames * height * width * sizeof(unsigned char));

        /* start time */
        GET_TIME(start);

        const double aspect_ratio = (double)width/height;
        double delta = Delta;

        dim3 threadsPerBlock(width, height);

        //figure out number of blocks, and threads per block
        fractal_kernel<<< num_frames, threadsPerBlock >>>(pic, width, height, num_frames, aspect_ratio, delta);

        hipDeviceSynchronize();

        /* end time */
        GET_TIME(end);
        double elapsed = end - start;
        printf("Parallel compute time: %.4f s\n", elapsed);

        /* write frames to BMP files */
        if ((width <= 320) && (num_frames <= 100)) { /* do not write if images large or many */
                for (int frame = 0; frame < num_frames; frame++) {
                        char name[32];
                        sprintf(name, "fractal%d.bmp", frame + 1000);
                        writeBMP(width, height, &pic[frame * height * width], name);
                }
        }

        hipFree(pic);
        return 0;
} /* main */
                                            